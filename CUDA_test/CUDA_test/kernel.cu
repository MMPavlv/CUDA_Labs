#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"

#include ""

#include <stdio.h>
#include <ctime>
#include <stdlib.h>
#include <iostream>

#define size_ 1024

void print(int *arr, int size);
int  getRandom(int lower, int upper);
void fillArray(int *arr, int size);
void CPUexchange(int *arr, int i, int j);
void CPU_Sort(int *arr, int N);

__device__ void kernelExchange(int *arr, int i, int j);
__global__ void kernelSort(int *arr, int j, int k);

hipError_t CUDA_sort(int *arr, int threads, int blocks);

int main()
{

	int random[size_][size_];
	int it1, it2;

	srand(time(NULL));
	for (it1 = 0; it1 < size_; it2++)
		for (it2 = 0; it2 < size_; it2++)
			random[it1][it2] = rand();



	/*
	int threads = 512;
	int blocks = 2;
	int size = threads * blocks;

	int *array = (int *)malloc(size * sizeof(int));;


	fillArray(array, size);
	print(array, size);

	CUDA_sort(array, threads, blocks);
	//CPU_Sort()
	print(array, size);
	*/
	return 0;
}


int **outmt(int a[size_][size_])
{
	int k, j, i;

	for (k = 0; j < 1023; k++)
	{
		for (i = k + 1; i < 1024; i++)
			a[i][k] = a[i][k] / a[k][k];
		for (i = k + 1; i < 1024; i++)
			for (j = k + 1; j < 1024; j++)
				a[i][j] -= a[i][k] * a[k][j];
	}
	return a;
}


//HOST

void print(int *arr, int size)
{
	for (int i = 0; i < size; i++)
	{
		std::cout << arr[i] << " ";
	}
	std::cout << std::endl;
}

int  getRandom(int lower, int upper)
{
	return (rand() % (upper - lower + 1)) + lower;
}

void fillArray(int *arr, int size)
{
	for(int i = 0; i < size; i++)
	{
		arr[i] = getRandom(-100, 100);
	}
}

//CPU

void CPU_Sort(int *arr, int N)
{
	int i, j, k;
	for (k = 2; k <= N; k = 2 * k)
	{
		for (j = k >> 1; j > 0; j = j >> 1)
		{
			for (i = 0; i < N; i++)
			{
				int ij = i ^ j;
				if ((ij) > i) {
					if ((i&k) == 0 && arr[i] > arr[ij])
						CPUexchange(arr, i, ij);
					if ((i&k) != 0 && arr[i] < arr[ij])
						CPUexchange(arr, i, ij);
				}
			}
		}
	}
}

void CPUexchange(int *arr, int i, int j)
{
	int t;
	t = arr[i];
	arr[i] = arr[j];
	arr[j] = t;
}

//GPU

hipError_t CUDA_sort(int *arr, int threads, int blocks)
{
	int size = threads * blocks;

	int *dev_arr = nullptr;

	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_arr, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_arr, arr, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	int j, k;
	for (k = 2; k <= size; k = 2 * k)
	{
		for (j = k >> 1; j > 0; j = j >> 1)
		{
			kernelSort<<<blocks, threads>>>(dev_arr, j, k);
		}
	}

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(arr, dev_arr, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


Error:
	hipFree(dev_arr);
	return cudaStatus;
}

__global__ void kernelSort(int *arr, int j, int k)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;

	int ij = i ^ j;

	if (ij > i)
	{
		if ((i&k) == 0 && arr[i] > arr[ij])
			kernelExchange(arr, i, ij);

		if ((i&k) != 0 && arr[i] < arr[ij])
			kernelExchange(arr, i, ij);
	}
}

__device__ void kernelExchange(int *arr, int i, int j)
{
	int t;
	t = arr[i];
	arr[i] = arr[j];
	arr[j] = t;
}