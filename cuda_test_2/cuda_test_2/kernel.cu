
#include "hip/hip_runtime.h"



#include <stdio.h>
#include <ctime>
#include <stdlib.h>
#include <iostream>

__constant__ unsigned long long globalN[1];

__global__ void kernel_isPerfectNumber(bool *arr);
__device__ bool isPerfectNumber(unsigned  long long number);
bool isPerfectNumber_(unsigned long long number);
void host_isPerfectNumber(bool *arr, unsigned long long size);
hipError_t CUDA_get_perfect_numbers(bool *arr, unsigned long long threads, unsigned long long blocks, unsigned long long N);

int main()
{

	unsigned long long threads = 512;
	unsigned long long blocks = 4;
	unsigned long long size = threads * blocks;
	unsigned long long N = size;
	bool *array_cpu = (bool *)malloc(size * sizeof(bool));
	bool *array_gpu = (bool *)malloc(size * sizeof(bool));

	unsigned int start_time;
	unsigned int end_time;
	unsigned int search_time;

	start_time = clock();
	host_isPerfectNumber(array_cpu, N);
	end_time = clock();
	search_time = end_time - start_time;
	std::cout << search_time / 1000.0 << std::endl;


	start_time = clock();
	CUDA_get_perfect_numbers(array_gpu, threads, blocks, N);
	end_time = clock();
	search_time = end_time - start_time;
	std::cout << search_time / 1000.0 << std::endl;

	for (unsigned long long i = 0; i < N; i++)
	{
		if (array_gpu[i])
			std::cout << i << std::endl;
	}

	return 0;
}

//CPU
void host_isPerfectNumber(bool *arr, unsigned long long size)
{
	for (unsigned long long i = 0; i < size; i++)
	{
		if (isPerfectNumber_(i))
			arr[i] = true;
		else
			arr[i] = false;
	}
}

bool isPerfectNumber_(unsigned long long number)
{
	unsigned long long i = 1, sum = 0;
	while (i < number)
	{
		if (number%i == 0)
			sum = sum + i;
		i++;
	}

	if (sum == number)
		return true;
	else
		return false;
}


//GPU

hipError_t CUDA_get_perfect_numbers(bool *arr, unsigned long long threads, unsigned long long blocks, unsigned long long N)
{
	unsigned long long size = threads * blocks;

	bool *dev_arr = nullptr;

	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_arr, size * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(globalN), &N, sizeof(unsigned long long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Memcpy failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	kernel_isPerfectNumber <<<blocks, threads >>> (dev_arr);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(arr, dev_arr, size * sizeof(bool), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


Error:
	hipFree(dev_arr);
	return cudaStatus;
}

__global__ void kernel_isPerfectNumber(bool *arr)
{
	unsigned long long i = threadIdx.x + blockDim.x*blockIdx.x;

	if (i >= globalN[0])
	{
		arr[i] = false;
		return;
	}

	if (isPerfectNumber(i))
		arr[i] = true;
	else
		arr[i] = false;
}

__device__ bool isPerfectNumber(unsigned long long number)
{
	unsigned long long i = 1, sum = 0;
	while (i < number)
	{
		if (number%i == 0)
			sum = sum + i;
		i++;
	}

	if (sum == number)
		return true;
	else
		return false;
}